#include "hip/hip_runtime.h"
#include <iostream>
#include <stdexcept>

#include "matrix/matrix_operations.hh"
#include "memory_checks/cuda_checks.hh"

__global__ void matAddKernel(
                             float *d_A,
                             float *d_B,
                             float *d_C,
                             int width,
                             int height
                            )
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col < width && row < height)
        d_C[row * width + col] = d_A[row * width + col] + d_B[row * width + col];
}

__global__ void matAddKernel(
                             float *d_A,
                             float B,
                             int width,
                             int height
                            )
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col < width && row < height)
        d_A[row * width + col] += B;
}

void computeDims(dim3& dimGrid, dim3& dimBlock, int width, int height)
{
    dimGrid.x = ceil(width / 16.0);
    dimGrid.y = ceil(height / 16.0);

    dimBlock.x = 16.0;
    dimBlock.y = 16.0;
}

Matrix matAdd(const Matrix& A, const Matrix& B)
{
    if (A.width != B.width || A.height != B.height)
        throw std::invalid_argument("Matrix width or height missmatch!");
    std::cout << "A[0] = " << A.data[0] << "\n";

    ERRCHECK(hipSetDevice(1));
    dim3 dimGrid;
    dim3 dimBlock;

    computeDims(dimGrid, dimBlock, A.width, A.height);

    int size = A.width * A.height * sizeof(float);

    float *d_A;    
    float *d_B;
    float *d_C;

    ERRCHECK(hipMalloc((void **) &d_A, size));
    ERRCHECK(hipMalloc((void **) &d_B, size));
    ERRCHECK(hipMalloc((void **) &d_C, size));
    std::cout << "Finished device malloc\n";

    ERRCHECK(hipMemcpy(d_A, A.data, size, hipMemcpyHostToDevice));
    std::cout << "Finished d_A memcpy\n";
    ERRCHECK(hipMemcpy(d_B, B.data, size, hipMemcpyHostToDevice));
    std::cout << "Finished d_B memcpy\n";

    std::cout << "Launching kernel matmat\n";
    matAddKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, A.width, A.height);
    ERRCHECK(hipGetLastError());
    std::cout << "Finished kernel matmat\n";

    float *C = static_cast<float *>(calloc(1, size));


    std::cout << "Begin Memcpy\n";
    ERRCHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));
    std::cout << "Finished Memcpy\n";

    ERRCHECK(hipFree(d_A));
    ERRCHECK(hipFree(d_B));
    ERRCHECK(hipFree(d_C));

    hipDeviceSynchronize();

    return Matrix(A.width, A.height, C);
}

Matrix& matAdd(Matrix& A, const float& B)
{
    ERRCHECK(hipSetDevice(1));
    dim3 dimGrid;
    dim3 dimBlock;

    computeDims(dimGrid, dimBlock, A.width, A.height);

    int size = A.width * A.height * sizeof(float);

    float *d_A;    

    ERRCHECK(hipMalloc((void **) &d_A, size));

    ERRCHECK(hipMemcpy(d_A, A.data, size, hipMemcpyHostToDevice));

    matAddKernel<<<dimGrid, dimBlock>>>(d_A, B, A.width, A.height);
    ERRCHECK(hipGetLastError());

    ERRCHECK(hipMemcpy(A.data, d_A, size, hipMemcpyDeviceToHost));

    ERRCHECK(hipFree(d_A));

    hipDeviceSynchronize();

    return A;
}
